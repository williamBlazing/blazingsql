#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cudf/utilities/error.hpp>
#include <exception>
#include "GPUManager.cuh"
#include "Utils.cuh"

namespace ral {
namespace config {

size_t gpuMemorySize() {
	int currentDeviceId = 0;
	struct hipDeviceProp_t props;
	CUDA_TRY( hipSetDevice(currentDeviceId) );
	hipGetDeviceProperties(&props, currentDeviceId);
	size_t free, total;
	hipMemGetInfo(&free, &total);

	return free;
}

}	// namespace config
}	// namespace ral
